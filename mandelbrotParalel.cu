//==============================================
//TRABALHO DE PROGRAMAÇÃO PARALELA E DISTRIBUÍDA
//		         Mandelbrot Set
//	                 CUDA
//          Daniela Kuinchtner, 152064
//==============================================


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

#define THREADSPERBLOCK 1024


__global__ void brot(char *d_A, int max_row, int max_column, int max_n, int n);


int main(int argc, char *argv[]){

	int max_row, max_column, max_n;

	max_row = atoi(argv[1]);
	max_column = atoi(argv[2]);
	max_n = atoi(argv[3]);

	int n = max_row * max_column;
	size_t size = n * sizeof(char);

	int nBlocks = (n+THREADSPERBLOCK-1) / THREADSPERBLOCK; 

	char *h_A; 
    char *d_A; 

    h_A = (char *)malloc(size);

    hipSetDevice(0); 

    hipMalloc((void**)&d_A, size); 

    hipMemcpy(d_A, h_A, size ,hipMemcpyHostToDevice);

    brot <<< nBlocks, THREADSPERBLOCK >>> (d_A, max_row, max_column, max_n, n);
	
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost); 
    
    int i = 0;
    for(int r = 0; r < max_row; ++r){
		for(int c = 0; c < max_column; ++c){

			cout << h_A[i++]; 
		}
		cout << "\n"; 
	}

    hipFree(d_A);
}


__global__ void brot(char *d_A, int max_row, int max_column, int max_n, int n){
        
	int k = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if(i < n){ 
    	int r = i / max_column;
    	int c =  i % max_column;

	    float x=0, y=0, tmp=0;
	    while((x*x + y*y) < 4 && ++k < max_n) {
	                tmp = x*x - y*y + ((float) c * 2 / max_column - 1.5);
	                y = x*y*2 + ((float) r * 2 / max_row - 1);
	                x = tmp;
	    }
	    d_A[i]=(k == max_n ? '#' : '.');		
    }
}